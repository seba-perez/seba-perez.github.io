#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------
   Copyright (C) 2016-2017  Miguel Carcamo, Pablo Roman, Simon Casassus,
   Victor Moral, Fernando Rannou - miguel.carcamo@usach.cl
   This program includes Numerical Recipes (NR) based routines whose
   copyright is held by the NR authors. If NR routines are included,
   you are required to comply with the licensing set forth there.
   Part of the program also relies on an an ANSI C library for multi-stream
   random number generation from the related Prentice-Hall textbook
   Discrete-Event Simulation: A First Course by Steve Park and Larry Leemis,
   for more information please contact leemis@math.wm.edu
   Additionally, this program uses some NVIDIA routines whose copyright is held
   by NVIDIA end user license agreement (EULA).
   For the original parts of this code, the following license applies:
   This program is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.
   This program is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.
   You should have received a copy of the GNU General Public License
   along with this program. If not, see <http://www.gnu.org/licenses/>.
 * -------------------------------------------------------------------------
 */

#include "frprmn.cuh"
#include "directioncosines.cuh"
#include "pillBox2D.cuh"
#include "gaussianSinc2D.cuh"
#include "gaussian2D.cuh"
#include "sinc2D.cuh"
#include "pswf_12D.cuh"
#include "fixedpoint.cuh"
#include <time.h>

int num_gpus;

inline bool IsAppBuiltAs64()
{
  #if defined(__x86_64) || defined(AMD64) || defined(_M_AMD64)
        return 1;
  #else
        return 0;
  #endif
}

/*
   This is a function that runs gpuvmem and calculates new regularization values according to the Belge et al. 2002 paper.
 */
std::vector<float> runGpuvmem(std::vector<float> args, Synthesizer *synthesizer)
{

        int cter = 0;
        std::vector<Fi*> fis = synthesizer->getOptimizator()->getObjectiveFuntion()->getFi();
        for(std::vector<Fi*>::iterator it = fis.begin(); it != fis.end(); it++)
        {
                if(cter)
                        (*it)->setPenalizationFactor(args[cter]);
                cter++;
        }

        synthesizer->clearRun();
        synthesizer->run();
        std::vector<float> fi_values = synthesizer->getOptimizator()->getObjectiveFuntion()->get_fi_values();
        std::vector<float> lambdas(fi_values.size(), 1.0f);

        for(int i=0; i < fi_values.size(); i++)
        {
                if(i>0)
                {
                        lambdas[i] = fi_values[0]/fi_values[i] * (logf(fi_values[i])/logf(fi_values[0]));
                        if(lambdas[i] < 0.0f)
                                lambdas[i] = 0.0f;
                }
        }

        return lambdas;
}

void optimizationOrder(Optimizator *optimizator, Image *image){
        optimizator->setImage(image);
        optimizator->setFlag(0);
        optimizator->optimize();
        /*optimizator->setFlag(1);
           optimizator->optimize();
           optimizator->setFlag(2);
           optimizator->optimize();
           optimizator->setFlag(3);
           optimizator->optimize();*/
}